#include "index_list_cuda.h"

#include <hip/hip_runtime.h>
#include <cub/device/device_select.cuh>
#include <cub/iterator/counting_input_iterator.cuh>

template<typename T>
struct ZeroCmp
{
	const T* conditions;
	const int startid;

	ZeroCmp(const int startid, const T* conditions) :
		startid(startid), conditions(conditions)
	{ }

	__device__ __host__ __forceinline__
	bool operator() (const int &id)
	{
	  return (conditions[ id - startid ] != 0);
	}
};

template <typename T>
static
void c_generate_index_list_cuda_generic_device(
			const T* dev_conditions,
			const int startid, const int endid,
			int* dev_indices,
			int* dev_nvalid, hipStream_t stream)
{
	static size_t storageSize = 0;
	static char* storage = nullptr;

	const int n = endid - startid + 1;

	// Argument is the offset of the first element
	hipcub::CountingInputIterator<int> iterator(startid);

	// Determine temporary device storage requirements
	size_t storageRequirement;
	hipcub::DeviceSelect::Flagged(nullptr, storageRequirement,
			iterator, dev_conditions, dev_indices,
			dev_nvalid, n, stream);

	// Allocate temporary storage (only if not enough)
	if (storageRequirement > storageSize)
	{
		hipFree(storage);
		hipMalloc(&storage, storageRequirement);
		storageSize = storageRequirement;
	}

	ZeroCmp<T> select(startid, dev_conditions);
	hipcub::DeviceSelect::If(storage, storageRequirement,
			iterator, dev_indices,
			dev_nvalid, n,
			select, stream);
}


template <typename T>
static
void c_generate_index_list_cuda_batched_generic(
			const int batch_size,
			const T* dev_conditions, const int cond_stride,
			const int startid, const int endid,
			int* dev_indices, const int idx_stride,
			int* dev_nvalid, hipStream_t stream)
{
	for (int i = 0; i < batch_size; i++)
		c_generate_index_list_cuda_generic_device(
				dev_conditions + cond_stride*i,
				startid, endid,
				dev_indices + idx_stride*i,
				dev_nvalid + i, stream);
}

template <typename T>
static
void c_generate_index_list_cuda_generic(
			const T* dev_conditions,
			const int startid, const int endid,
			int* dev_indices,
			int& nvalid, hipStream_t stream)
{
	static int* dev_nvalid = nullptr;
	if (dev_nvalid == nullptr)
			hipMalloc(&dev_nvalid, sizeof(int));

	c_generate_index_list_cuda_generic_device(
			dev_conditions, startid, endid, dev_indices, dev_nvalid, stream);

	hipMemcpyAsync(&nvalid, dev_nvalid, sizeof(int), hipMemcpyDeviceToHost, stream);
	hipStreamSynchronize(stream);
}

///
/// Exposed functions
/// 
/// Non-batched first
/// 

void c_generate_index_list_cuda_i1(
			const char* dev_conditions,
			const int startid, const int endid,
			int* dev_indices,
			int& nvalid, hipStream_t stream)
{
	c_generate_index_list_cuda_generic(dev_conditions, startid, endid, dev_indices, nvalid, stream);
}

void c_generate_index_list_cuda_i4(
			const int* dev_conditions,
			const int startid, const int endid,
			int* dev_indices,
			int& nvalid, hipStream_t stream)
{
	c_generate_index_list_cuda_generic(dev_conditions, startid, endid, dev_indices, nvalid, stream);
}

/// 
/// And now batched
/// 

void c_generate_index_list_cuda_batched_i1(
	const int batch_size,
	const char* dev_conditions, const int cond_stride,
	const int startid, const int endid,
	int* dev_indices, const int idx_stride,
	int* dev_nvalid, hipStream_t stream)
{
c_generate_index_list_cuda_batched_generic(
		batch_size,
		dev_conditions, cond_stride,
		startid, endid,
		dev_indices, idx_stride,
		dev_nvalid, stream);
}

void c_generate_index_list_cuda_batched_i4(
		const int batch_size,
		const int* dev_conditions, const int cond_stride,
		const int startid, const int endid,
		int* dev_indices, const int idx_stride,
		int* dev_nvalid, hipStream_t stream)
{
	c_generate_index_list_cuda_batched_generic(
			batch_size,
			dev_conditions, cond_stride,
			startid, endid,
			dev_indices, idx_stride,
			dev_nvalid, stream);
}

