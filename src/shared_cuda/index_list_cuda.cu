#include "index_list_cuda.h"

#include <hip/hip_runtime.h>
#include <cub/device/device_select.cuh>
#include <cub/iterator/counting_input_iterator.cuh>

void c_generate_index_list_cuda_i1(
			const char* dev_conditions,
			const int startid, const int endid,
			int* dev_indices,
			int& nvalid, hipStream_t stream)
{
	static size_t storageSize = 0;
	static char* storage = nullptr;
	static int*  dev_nvalid = nullptr;

	const int n = endid - startid + 1;

	if (dev_nvalid == nullptr)
		hipMalloc(&dev_nvalid, sizeof(int));

	// Argument is the offset of the first element
	hipcub::CountingInputIterator<int> iterator(startid);

	// Determine temporary device storage requirements
	size_t storageRequirement;
	hipcub::DeviceSelect::Flagged(nullptr, storageRequirement,
			iterator, dev_conditions, dev_indices,
			dev_nvalid, n, stream);

	// Allocate temporary storage (only if not enough)
	if (storageRequirement > storageSize)
	{
		hipFree(storage);
		hipMalloc(&storage, storageRequirement);
		storageSize = storageRequirement;
	}

	hipcub::DeviceSelect::Flagged(storage, storageRequirement,
			iterator, dev_conditions, dev_indices,
			dev_nvalid, n, stream);

	hipMemcpyAsync(&nvalid, dev_nvalid, sizeof(int), hipMemcpyDeviceToHost, stream);
	hipStreamSynchronize(stream);
}



void c_generate_index_list_cuda_batched_i1(
			const char* dev_conditions,
			int nbatches,
			const int startid, const int endid,
			int* dev_indices,
			int& nvalid, hipStream_t stream)
{
	return;
}

